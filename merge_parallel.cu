#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAX_VAL 1000
#define BLOCK_SIZE 256

__device__ void mergeSort(int *input, int p, int r){
    if ( p < r ){
        if(input[p] > input[r]) {
            // swap elements
            int temp = input[p];
            input[p] = input[r];
            input[r] = temp;
        }
    }
}

__global__ void kernel(int *input, int size){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx * 2 < size) {
        mergeSort(input, idx * 2, idx * 2 + 1);
    }
}

int main(int argc, char* argv[]){
    int *d_input;
    int size;
    sscanf(argv[1], "%d", &size); // get array size from command line
    size *= sizeof(int);
    
    int *input = (int*)malloc(size);
    for(int i = 0; i < size/sizeof(int); i++){
        input[i] = rand() % MAX_VAL;
    }
    
    hipMalloc((void**)&d_input, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Timer start
    float gpu_time = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    kernel<<<(size + BLOCK_SIZE - 1) / (2*BLOCK_SIZE), BLOCK_SIZE>>>(d_input, size/sizeof(int));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    
    hipMemcpy(input, d_input, size, hipMemcpyDeviceToHost);
    
    printf("Elapsed time: %f s\n", gpu_time / 1000.0);
    
    hipFree(d_input);
    free(input);
    
    return 0;
}
